#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "roi_align_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                              \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
         i += blockDim.x * gridDim.x)

    /*** Forward ***/

    __device__ float bilinear_interpolate(const float* bottom_data, const int height, const int width,
                                          float y, float x, const int index /* index for debug only*/) {
            // deal with cases that inverse elements are out of feature map boundary
            if (y < -1.0 || y > height || x < -1.0 || x > width) {
                // empty
                return 0;
            }

            if (y <= 0) {
                y = 0;
            }
            if (x <= 0) {
                x = 0;
            }
            
            int y_low = (int)y;
            int x_low = (int)x;
            int y_high;
            int x_high;
            
            if (y_low >= height - 1) {
                y_high = y_low = height - 1;
                y = (float)y_low;
            } else {
                y_high = y_low + 1;
            }
            
            if (x_low >= width - 1) {
                x_high = x_low = width - 1;
                x = (float)x_low;
            } else {
                x_high = x_low + 1;
            }
            
            float ly = y - y_low;
            float lx = x - x_low;
            float hy = 1. -ly, hx = 1. - lx;
            // do bilinear interpolation
            float v1 = bottom_data[y_low * width + x_low];
            float v2 = bottom_data[y_low * width + x_high];
            float v3 = bottom_data[y_high * width + x_low];
            float v4 = bottom_data[y_high * width + x_high];
            float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

            float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

            return val;
        }

    __global__ void ROIAlignForward(const int nthreads, const float* bottom_data, const float spatial_scale, const int height, const int width,
                                    const int channels, const int aligned_height, const int aligned_width, const int sampling_ratio,
                                    const float* bottom_rois, float* top_data) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
            // (n, c, ph, pw) is an element in the aligned output
            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
            int c  = (index / aligned_width / aligned_height) % channels;
            int n  = index / aligned_width / aligned_height / channels;

            const float* offset_bottom_rois = bottom_rois + n * 5;
            int roi_batch_ind = offset_bottom_rois[0];

            // Do not using rounding; this implementation detail is critical
            float roi_start_w = offset_bottom_rois[1] * spatial_scale;
            float roi_start_h = offset_bottom_rois[2] * spatial_scale;
            float roi_end_w = offset_bottom_rois[3] * spatial_scale;
            float roi_end_h = offset_bottom_rois[4] * spatial_scale;

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w, 1.f);
            float roi_height = fmaxf(roi_end_h - roi_start_h, 1.f);
            float bin_size_h = roi_height / aligned_height;
            float bin_size_w = roi_width / aligned_width;

            const float* offset_bottom_data =
                bottom_data + (roi_batch_ind * channels + c) * height * width;

            // We use roi_bin_grid to sample the grid and mimic integral
            int roi_bin_grid_h = (sampling_ratio > 0)
                ? sampling_ratio
                : ceil(roi_height / aligned_height); // e.g., = 2
            int roi_bin_grid_w =
                (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / aligned_width);

            // We do average (integral) pooling inside a bin
            const float count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

            float output_val = 0.;
            for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
            {
                const float y = roi_start_h + ph * bin_size_h +
                    (iy + .5f) * bin_size_h / roi_bin_grid_h;  // e.g., 0.5, 1.5
                for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                    const float x = roi_start_w + pw * bin_size_w +
                    (ix + .5f) * bin_size_w / roi_bin_grid_w;

                    float val = bilinear_interpolate(
                        offset_bottom_data, height, width, y, x, index);
                    output_val += val;
                }
            }
            output_val /= count;

            top_data[index] = output_val;
        }
    }

    int ROIAlignForwardLaucher(const float* bottom_data, const float spatial_scale, const int num_rois, const int height, const int width,
                               const int channels, const int aligned_height, const int aligned_width,  const int sampling_ratio,
                               const float* bottom_rois, float* top_data, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * aligned_height * aligned_width * channels;
        hipError_t err;


        ROIAlignForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
          output_size, bottom_data, spatial_scale, height, width, channels,
          aligned_height, aligned_width, sampling_ratio, bottom_rois, top_data);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }

    /*** Backward ***/
    inline __device__ float gpu_atomic_add(const float val, float* address);
    inline __device__ float gpu_atomic_add(const float val, float* address) {
        return atomicAdd(address, val);
    }

    __device__ void bilinear_interpolate_gradient(const int height, const int width, float y, float x,
                                                  float& w1, float& w2, float& w3, float& w4,
                                                  int& x_low, int& x_high, int& y_low, int& y_high,
                                                  const int index /* index for debug only*/) {
        // deal with cases that inverse elements are out of feature map boundary
        if (y < -1.0 || y > height || x < -1.0 || x > width) {
            // empty
            w1 = w2 = w3 = w4 = 0.;
            x_low = x_high = y_low = y_high = -1;
            return;
        }

        if (y <= 0) {
            y = 0;
        }
        if (x <= 0) {
            x = 0;
        }

        y_low = (int)y;
        x_low = (int)x;

        if (y_low >= height - 1) {
            y_high = y_low = height - 1;
            y = (float)y_low;
        } else {
            y_high = y_low + 1;
        }

        if (x_low >= width - 1) {
            x_high = x_low = width - 1;
            x = (float)x_low;
        } else {
            x_high = x_low + 1;
        }

        float ly = y - y_low;
        float lx = x - x_low;
        float hy = 1. - ly, hx = 1. - lx;

        w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

        return;
    }

    __global__ void ROIAlignBackward(const int nthreads, const float* top_diff, const float spatial_scale, const int height, const int width,
                                     const int channels, const int aligned_height, const int aligned_width, const int sampling_ratio,
                                     float* bottom_diff, const float* bottom_rois) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
            // (n, c, ph, pw) is an element in the aligned output
            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
            int c  = (index / aligned_width / aligned_height) % channels;
            int n  = index / aligned_width / aligned_height / channels;

            const float* offset_bottom_rois = bottom_rois + n * 5;
            int roi_batch_ind = offset_bottom_rois[0];

            // Do not using rounding; this implementation detail is critical
            float roi_start_w = offset_bottom_rois[1] * spatial_scale;
            float roi_start_h = offset_bottom_rois[2] * spatial_scale;
            float roi_end_w = offset_bottom_rois[3] * spatial_scale;
            float roi_end_h = offset_bottom_rois[4] * spatial_scale;

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w, 1.f);
            float roi_height = fmaxf(roi_end_h - roi_start_h, 1.f);
            float bin_size_h = roi_height / aligned_height;
            float bin_size_w = roi_width / aligned_width;

            float* offset_bottom_diff =
                bottom_diff + (roi_batch_ind * channels + c) * height * width;

            int top_offset = (n * channels + c) * aligned_height * aligned_width;
            const float* offset_top_diff = top_diff + top_offset;
            const float top_diff_this_bin = offset_top_diff[ph * aligned_width + pw];

            // We use roi_bin_grid to sample the grid and mimic integral
            int roi_bin_grid_h = (sampling_ratio > 0)
                ? sampling_ratio
                : ceil(roi_height / aligned_height); // e.g., = 2
            int roi_bin_grid_w =
                (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / aligned_width);

            // We do average (integral) pooling inside a bin
            const float count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

            for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
            {
                const float y = roi_start_h + ph * bin_size_h +
                    (iy + .5f) * bin_size_h / roi_bin_grid_h; // e.g., 0.5, 1.5
                for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                    const float x = roi_start_w + pw * bin_size_w +
                        (ix + .5f) * bin_size_w / roi_bin_grid_w;

                    float w1, w2, w3, w4;
                    int x_low, x_high, y_low, y_high;

                    bilinear_interpolate_gradient(
                        height, width, y, x, w1, w2, w3, w4,
                        x_low, x_high, y_low, y_high, index);

                    float g1 = top_diff_this_bin * w1 / count;
                    float g2 = top_diff_this_bin * w2 / count;
                    float g3 = top_diff_this_bin * w3 / count;
                    float g4 = top_diff_this_bin * w4 / count;

                    if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
                        // atomicAdd(offset_bottom_diff + y_low * width + x_low, g1);
                        // atomicAdd(offset_bottom_diff + y_low * width + x_high, g2);
                        // atomicAdd(offset_bottom_diff + y_high * width + x_low, g3);
                        // atomicAdd(offset_bottom_diff + y_high * width + x_high, g4);
                        gpu_atomic_add(g1, offset_bottom_diff + y_low * width + x_low);
                        gpu_atomic_add(g2, offset_bottom_diff + y_low * width + x_high);
                        gpu_atomic_add(g3, offset_bottom_diff + y_high * width + x_low);
                        gpu_atomic_add(g4, offset_bottom_diff + y_high * width + x_high);
                    } // if
                } // ix
            } // iy
        } // CUDA_1D_KERNEL_LOOP
    } // RoIAlignBackward

    int ROIAlignBackwardLaucher(const float* top_diff, const float spatial_scale, const int batch_size, const int num_rois, const int height, const int width,
                                const int channels, const int aligned_height, const int aligned_width, const int sampling_ratio,
                                const float* bottom_rois, float* bottom_diff, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * aligned_height * aligned_width * channels;
        hipError_t err;

        ROIAlignBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
          output_size, top_diff, spatial_scale, height, width, channels,
          aligned_height, aligned_width,  sampling_ratio, bottom_diff, bottom_rois);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


#ifdef __cplusplus
}
#endif
